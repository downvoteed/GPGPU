#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include <stdio.h>

__global__ void helloCUDA(float f)
{
	printf("Hello thread %d, f=%f\n", threadIdx.x, f);
}

__device__ __host__ uint8_t calculateLBP(uchar3* image, int idx, int idy, int width, int height) {
    float center = 0.2989f * image[idy * width + idx].x + 0.5870f * image[idy * width + idx].y + 0.1140f * image[idy * width + idx].z;

    uint8_t lbp = 0;

    int directions[8][2] = { {-1,-1}, {0,-1}, {1,-1}, {1,0}, {1,1}, {0,1}, {-1,1}, {-1,0} };

    for (int i = 0; i < 8; i++) {
        int x = min(max(idx + directions[i][0], 0), width - 1);
        int y = min(max(idy + directions[i][1], 0), height - 1);

        float value = 0.2989f * image[y * width + x].x + 0.5870f * image[y * width + x].y + 0.1140f * image[y * width + x].z;

        lbp = (lbp << 1) | (value < center);
    }

    return lbp;
}

__device__ float compare(uint8_t lbp1, uint8_t lbp2) {
    uint8_t vector = ~(lbp1 ^ lbp2);
    return __popc(vector) / 8.0f;
}

__global__ void fuzzy_integral_and_LBP(uchar3* image1, uchar3* image2, uint8_t* lbpBackground, float* result, int width, int height) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx >= width || idy >= height) return;

    uchar3 pixel1 = image1[idy * width + idx];
    uchar3 pixel2 = image2[idy * width + idx];

    float r_ratio, g_ratio;
    if (pixel1.x > pixel2.x) {
        r_ratio = (float)pixel2.x / (float)pixel1.x;
    } else {
        r_ratio = (float)pixel1.x / (float)pixel2.x;
    }

    if (pixel1.y > pixel2.y) {
        g_ratio = (float)pixel2.y / (float)pixel1.y;
    } else {
        g_ratio = (float)pixel1.y / (float)pixel2.y;
    }

    // print pixels values
     //printf("%d %d %d %d %d %d\n", pixel1.x, pixel1.y, pixel1.z, pixel2.x, pixel2.y, pixel2.z);

    uint8_t lbp1 = lbpBackground[idy * width + idx];
    uint8_t lbp2 = calculateLBP(image2, idx, idy, width, height);

    float lbp_similarity = compare(lbp1, lbp2);

    float coefficients[] = { r_ratio, g_ratio, lbp_similarity };
    float temp;

    for(int i=0; i<2; ++i){
        for(int j=0; j<2-i; ++j){
            if(coefficients[j] > coefficients[j+1]){
                temp = coefficients[j];
                coefficients[j] = coefficients[j+1];
                coefficients[j+1] = temp;
            }
        }
    }

    float final_result = coefficients[0] * 0.1f + coefficients[1] * 0.3f + coefficients[2] * 0.6f;
   // printf("%f %f %f %f\n", coefficients[0], coefficients[1], coefficients[2], final_result);

    result[idy * width + idx] = final_result > 0.67f ? 0 : 255;
}


int main(int argc, char** argv)
{
    std::string input_path = "C:\\Users\\mouis\\EPITA\\GPGPU\\gpgpu\\dataset\\video.avi";
    std::string output_path = "C:\\Users\\mouis\\EPITA\\GPGPU\\gpgpu\\dataset\\outputGPU.mp4";

    std::vector<cv::Mat> frames;
    cv::VideoCapture cap(input_path);

    if (!cap.isOpened())
    {
        std::cerr << "Unable to open the file" << std::endl;
        return -1;
    }

    cv::Mat frame;
    cv::Size frameSize;
	while (cap.read(frame))
	{
		frameSize = frame.size();
		frames.push_back(frame.clone());
	}


    int fourcc = cv::VideoWriter::fourcc('H', '2', '6', '4');
    double fps = 30.0;
    cv::VideoWriter writer(output_path, fourcc, fps, frameSize, true);

    if (!writer.isOpened())
    {
        std::cerr << "Unable to open the output file for writing" << std::endl;
        return -1;
    }

    int width = frames[0].cols;
    int height = frames[0].rows;
    uchar3* d_image1;
    uchar3* d_image2;
    float* d_result;
    hipMalloc(&d_image1, width * height * sizeof(uchar3));
    hipMalloc(&d_image2, width * height * sizeof(uchar3));
    hipMalloc(&d_result, width * height * sizeof(float));

    // Calculate and copy LBP of the first frame to the GPU
    cv::Mat firstFrame = frames[0];
    uchar3* h_image1 = firstFrame.ptr<uchar3>();
    uint8_t* h_lbpBackground = new uint8_t[width * height];
    for(int y=0; y<height; ++y){
        for(int x=0; x<width; ++x){
            h_lbpBackground[y * width + x] = calculateLBP(h_image1, x, y, width, height);
        }
    }
    uint8_t* d_lbpBackground;
    hipMalloc(&d_lbpBackground, width * height * sizeof(uint8_t));
    hipMemcpy(d_lbpBackground, h_lbpBackground, width * height * sizeof(uint8_t), hipMemcpyHostToDevice);

    for (size_t i = 1; i < frames.size(); i++)
    {
        hipMemcpy(d_image1, firstFrame.ptr<uchar3>(), width * height * sizeof(uchar3), hipMemcpyHostToDevice);
        hipMemcpy(d_image2, frames[i].ptr<uchar3>(), width * height * sizeof(uchar3), hipMemcpyHostToDevice);

        dim3 blockSize(16, 16);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
        fuzzy_integral_and_LBP<<<gridSize, blockSize>>>(d_image1, d_image2, d_lbpBackground, d_result, width, height);

        cv::Mat processed_frame(height, width, CV_32F);
        hipMemcpy(processed_frame.ptr<float>(), d_result, width * height * sizeof(float), hipMemcpyDeviceToHost);

        processed_frame.convertTo(processed_frame, CV_8UC1, 255.0);
        cv::cvtColor(processed_frame, processed_frame, cv::COLOR_GRAY2BGR);
        writer.write(processed_frame);
    }

    hipFree(d_image1);
    hipFree(d_image2);
    hipFree(d_result);
    hipFree(d_lbpBackground);

    cap.release();
    writer.release();

    return 0;
}
