#include "hip/hip_runtime.h"
#include "frame-helper.cuh"
#include "segmentation-helper.cuh"
#include <chrono>

void process_frames(const std::string& input_path, const std::string& output_path) {
    cv::VideoCapture cap(input_path);
    if (!cap.isOpened()) {
        std::cerr << "Unable to open the file" << std::endl;
        exit(1);
    }

    cv::Mat frame;
    cap.read(frame);
    cv::Size frameSize = frame.size();

    int fourcc = cv::VideoWriter::fourcc('H', '2', '6', '4');
    double fps = 30.0;
    cv::VideoWriter writer(output_path, fourcc, fps, frameSize, true);

    if (!writer.isOpened()) {
        std::cerr << "Unable to open the output file for writing" << std::endl;
        exit(1);
    }

    int width = frame.cols;
    int height = frame.rows;
    uchar3* d_image1;
    uchar3* d_image2;
    uint8_t* d_lbpBackground;
    float* d_result;
    hipMalloc(&d_image1, width * height * sizeof(uchar3));
    hipMalloc(&d_image2, width * height * sizeof(uchar3));
    hipMalloc(&d_lbpBackground, width * height * sizeof(uint8_t));
    hipMalloc(&d_result, width * height * sizeof(float));
    hipMemcpy(d_image1, frame.ptr<uchar3>(), width * height * sizeof(uchar3), hipMemcpyHostToDevice);

    uint8_t* h_lbpBackground = new uint8_t[width * height];
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            h_lbpBackground[y * width + x] = calculateLBP(frame.ptr<uchar3>(), x, y, width, height);
        }
    }

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipMemcpyAsync(d_lbpBackground, h_lbpBackground, width * height * sizeof(uint8_t), hipMemcpyHostToDevice, stream1);

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    int frameCount = 0;
    auto start = std::chrono::high_resolution_clock::now();
    bool isFrameRead = true;

    do {
        frameCount++;
        if (frameCount % 100 == 0) {
            auto now = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::seconds>(now - start);
            std::cout << "Frames per second: " << static_cast<double>(frameCount) / duration.count() << std::endl;
        }

        hipMemcpyAsync(d_image2, frame.ptr<uchar3>(), width * height * sizeof(uchar3), hipMemcpyHostToDevice, stream1);
        fuzzy_integral<<<gridSize, blockSize, 0, stream1>>>(d_image1, d_image2, d_lbpBackground, d_result, width, height);

        if (isFrameRead) {
            isFrameRead = cap.read(frame);
            hipMemcpyAsync(d_image1, d_image2, width * height * sizeof(uchar3), hipMemcpyDeviceToDevice, stream2);
        }

        cv::Mat processed_frame(height, width, CV_32F);
        hipMemcpyAsync(processed_frame.ptr<float>(), d_result, width * height * sizeof(float), hipMemcpyDeviceToHost, stream1);
        hipStreamSynchronize(stream1);

        cv::Mat output_frame;
        processed_frame.convertTo(output_frame, CV_8UC1, 255.0);
        writer.write(output_frame);
    } while (isFrameRead);

    delete[] h_lbpBackground;
    hipFree(d_image1);
    hipFree(d_image2);
    hipFree(d_lbpBackground);
    hipFree(d_result);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    cap.release();
    writer.release();
}
