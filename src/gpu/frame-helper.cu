#include "hip/hip_runtime.h"
#include "frame-helper.cuh"
#include "segmentation-helper.cuh"

void process_frames(const std::string& input_path, const std::string& output_path) {
	std::vector<cv::Mat> frames;
	cv::VideoCapture cap(input_path);

	if (!cap.isOpened())
	{
		std::cerr << "Unable to open the file" << std::endl;
		exit(1);
	}

	cv::Mat frame;
	cv::Size frameSize;
	while (cap.read(frame))
	{
		frameSize = frame.size();
		frames.push_back(frame.clone());
	}


	int fourcc = cv::VideoWriter::fourcc('H', '2', '6', '4');
	double fps = 30.0;
	cv::VideoWriter writer(output_path, fourcc, fps, frameSize, true);

	if (!writer.isOpened())
	{
		std::cerr << "Unable to open the output file for writing" << std::endl;
		exit(1);
	}

	int width = frames[0].cols;
	int height = frames[0].rows;
	uchar3* d_image1;
	uchar3* d_image2;
	float* d_result;
	hipMalloc(&d_image1, width * height * sizeof(uchar3));
	hipMalloc(&d_image2, width * height * sizeof(uchar3));
	hipMalloc(&d_result, width * height * sizeof(float));

	// Calculate and copy LBP of the first frame to the GPU
	cv::Mat firstFrame = frames[0];
	uchar3* h_image1 = firstFrame.ptr<uchar3>();
	uint8_t* h_lbpBackground = new uint8_t[width * height];
	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			h_lbpBackground[y * width + x] = calculateLBP(h_image1, x, y, width, height);
		}
	}
	uint8_t* d_lbpBackground;
	hipMalloc(&d_lbpBackground, width * height * sizeof(uint8_t));
	hipMemcpy(d_lbpBackground, h_lbpBackground, width * height * sizeof(uint8_t), hipMemcpyHostToDevice);

	for (size_t i = 1; i < frames.size(); i++)
	{
		hipMemcpy(d_image1, firstFrame.ptr<uchar3>(), width * height * sizeof(uchar3), hipMemcpyHostToDevice);
		hipMemcpy(d_image2, frames[i].ptr<uchar3>(), width * height * sizeof(uchar3), hipMemcpyHostToDevice);

		dim3 blockSize(32, 32);
		dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
		fuzzy_integral<< <gridSize, blockSize >> > (d_image1, d_image2, d_lbpBackground, d_result, width, height);

		cv::Mat processed_frame(height, width, CV_32F);
		hipMemcpy(processed_frame.ptr<float>(), d_result, width * height * sizeof(float), hipMemcpyDeviceToHost);

		processed_frame.convertTo(processed_frame, CV_8UC1, 255.0);
		cv::cvtColor(processed_frame, processed_frame, cv::COLOR_GRAY2BGR);
		writer.write(processed_frame);
	}

	hipFree(d_image1);
	hipFree(d_image2);
	hipFree(d_result);
	hipFree(d_lbpBackground);

	cap.release();
	writer.release();
}
